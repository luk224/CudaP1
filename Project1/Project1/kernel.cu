#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void vectorAddKernel(float *a, float *b, float *c, int n) {
	// Escribir en c la suma de a y b
};


void vectorAdd(float *a, float *b, float *c, int n) {
	size_t size = n * sizeof(float);

	float *d_a = NULL;
	float *d_b = NULL;
	float *d_c = NULL;

	//malloc en cuda para d_a, d_b y d_c

	//copiar los datos de a y b a d_a y d_b respectivamente

	//llamada al kernel
	vectorAddKernel<<<1, 1 >>>(d_a, d_b, d_c, n);

	//copiar los datos de vuelta. Los datos calculados y guardados en d_c han de ser copiados a c.

	//liberar memoria que ya no se necesita
};
